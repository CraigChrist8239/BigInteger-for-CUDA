#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <cstring>
#include <cmath>
#include <ctime>

class BigInteger {
public: 
	enum Sign { negative = -1, zero = 0, positive = 1 };

	Sign sign;
	unsigned int totalBlockCount;
	unsigned int * blocks;

public:
#pragma region Utilities
	__device__ __host__ BigInteger * sqrt() {
		BigInteger * a = BigInteger::toValue((unsigned __int64)1);
		BigInteger * b = MakeACopy();
		*b >>= 5;
 		*b += 8;

		int loopCount = 0;
		while(*b >= *a) {
			BigInteger* mid = a->MakeACopy();
			*mid += *b;
			*mid >>= 1;

			BigInteger* midSqr = mid->MakeACopy();
			*midSqr *= *mid;
			if(*midSqr > *this) {
				*mid -= 1;
				*b = *mid;
			}
			else {
				*mid += 1;
				*a = *mid;
			}

			free(mid->blocks);
			free(mid);
			free(midSqr->blocks);
			free(midSqr);
			loopCount++;
		}

		*a -= 1;
		*this = *a;

		free(a->blocks);
		free(a);
		free(b->blocks);
		free(b);

		return this;
	}

	__device__ __host__ void toString(char * buffer) {
		if(sign == zero) {
			*(buffer++) = '0';
			*buffer = 0;
		}
		else {
			bool is_negative = sign == negative;
			if(is_negative) {
				-(*this);
				*(buffer++) = '-';
			}

			BigInteger * a = toValue((unsigned __int64)1, totalBlockCount);

			while(true) {
				*a *= 10;

				if(*a > *this) {
					*a /= 10;
					break;
				}
			}
		
			BigInteger * b = MakeACopy();
			while(*a >= 10) {
				BigInteger * c = a->MakeACopy();
				BigInteger * d = b->MakeACopy();

				*d /= *c;
				*b = *c;
				free(c->blocks);
				free(c);
				*a /= 10;

				*(buffer++) = '0' + d->blocks[0];
				free(d->blocks);
				free(d);
			}
		
			free(a->blocks);
			free(a);
			*(buffer++) = '0' + b->blocks[0];
			free(b->blocks);
			free(b);
			*buffer = 0;
			if(is_negative) -(*this);
		}
	}

	__device__ __host__ void IsZeroUpdate() {
		bool isZero = true;
		for(int i = 0; i < totalBlockCount; i++) {
			if(blocks[i] != 0) { isZero = false; break; }
		}
		if(isZero) sign = zero;
	}

	__device__ __host__ BigInteger& operator <<=(unsigned int x) {
		if(sign == zero || x == 0) { }
		else {
			if(x/32 > 0) {
				IncreaseIntegerSize(x / 32);

				for(int i = totalBlockCount - 1; i >= (x / 32); i--)
					blocks[i] = blocks[i-(x/32)];
			}
			unsigned int carry = 0;
			for(int i = 0; i < (x / 32); i++) blocks[i] = 0;
			for(int i = 0; i < totalBlockCount; i++) {
				unsigned __int64 thisBlock = (unsigned __int64)blocks[i] << (x % 32);
				blocks[i] = (unsigned int)thisBlock | carry;
				carry = (unsigned int)(thisBlock >> 32);
			}

			if(carry) {
				IncreaseIntegerSize(1);
				blocks[totalBlockCount - 1] = carry;
			}
		}

		return *this;
	}

	__device__ __host__ BigInteger& operator >>=(unsigned int x) {
		if(sign == zero || x == 0) { }
		else if(x >= 32*(totalBlockCount-1))
			*this = 0;
		else {
			int i = 0;
			for(; i < totalBlockCount - (x/32); i++)
				blocks[i] = blocks[i+(x/32)];
			unsigned int carry = 0;
			for(; i < totalBlockCount; i++) blocks[i] = 0;
			for(int i = totalBlockCount-1; i >= 0; i--) {
				unsigned __int64 thisBlock = ((unsigned __int64)blocks[i] << 32) >> (x % 32);
				blocks[i] = (unsigned int)(thisBlock >> 32) | carry;
				carry = (unsigned int)thisBlock;
			}
		}

		return *this;
	}

	__device__ __host__ void operator -() {
		if(sign == negative) sign = positive;
		else if(sign == positive) sign = negative;
	}

	__device__ __host__ void ResizeInteger(unsigned int newBlockCount) {
		unsigned int * old_blocks = blocks;
		blocks = (unsigned int*)malloc(newBlockCount*sizeof(unsigned int));
		memset(blocks, 0, newBlockCount*sizeof(unsigned int));
		memcpy(blocks, old_blocks, ((newBlockCount > totalBlockCount) ? totalBlockCount : newBlockCount) * sizeof(unsigned int));
		
		totalBlockCount = newBlockCount;
		free(old_blocks);
	}
	
	__device__ __host__ void SetBitAtOffset(unsigned int offset) {
		if(totalBlockCount < ((offset+1) / 32) + (((offset+1) % 32) ? 1 : 0))
			ResizeInteger(((offset+1) / 32) + (((offset+1) % 32) ? 1 : 0));

		if(sign == zero) sign = positive;

		blocks[offset / 32] |= (1 << (offset % 32));
	}

	__device__ __host__ void IncreaseIntegerSize(unsigned int additional_block_count) {
		ResizeInteger(totalBlockCount + additional_block_count);
	}

	__device__ __host__ BigInteger* MakeACopy() {
		BigInteger*out = (BigInteger*)malloc(sizeof(BigInteger));
		out->blocks = (unsigned int*)malloc(totalBlockCount * sizeof(unsigned int));
		out->totalBlockCount = totalBlockCount;
		out->sign = sign;
		memset(out->blocks, 0, totalBlockCount);
		memcpy(out->blocks, blocks, totalBlockCount * sizeof(unsigned int));
		return out;
	}

	__device__ __host__ static BigInteger* toValue(char * value, unsigned int block_count) {
		BigInteger * out = toValue((unsigned __int64)0, block_count);
		bool outNeg = (*value == '-');
		if(outNeg) value++;
		unsigned int digitCount = strlen(value);
		BigInteger* a = toValue((unsigned __int64)1, block_count);
		for(int i = 0; i < digitCount - 1; i++)
			*a *= 10;
		while(*a > 1) {
			BigInteger * b = a->MakeACopy();
			*b *= (*(value++) - '0');
			*out += *b;
			free(b->blocks);
			free(b);
			*a /= 10;
		}
		*a *= (*value - '0');
		*out += *a;
		free(a->blocks);
		free(a);

		if(outNeg) out->sign = negative;
		out->IsZeroUpdate();
		return out;
	}

	__device__ __host__ static BigInteger* toValue(Sign sign, unsigned int * digits, unsigned int block_count) {
		BigInteger * out = (BigInteger*)malloc(sizeof(BigInteger));
		out->blocks = (unsigned int*)malloc(block_count*sizeof(unsigned int));
		memcpy(out->blocks, digits, block_count*sizeof(unsigned int));
		out->totalBlockCount = block_count;
		out->sign = sign;
		return out;
	}

	__device__ __host__ static BigInteger* toValue(__int64 value, unsigned int block_count) {
		if(block_count < 2) block_count = 2;

		BigInteger * out = (BigInteger*)malloc(sizeof(BigInteger));
		out->blocks = (unsigned int*)malloc(block_count*sizeof(unsigned int));
		memset(out->blocks, 0, block_count*sizeof(unsigned int));
		out->sign = positive;
		if(value == 0) 
			out->sign = zero;
		else if(value < 0) {
			out->sign = negative;
			value = -value;
		}
		
		out->totalBlockCount = block_count;
		out->blocks[0] = value & 0xFFFFFFFF;
		out->blocks[1] = (value >> 32) & 0xFFFFFFFF;
		return out;
	}

	__device__ __host__ static BigInteger* toValue(__int64 value) {
		return toValue(value, 2);
	}
	
	__device__ __host__ static BigInteger* toValue(unsigned __int64 value, unsigned int block_count) {
		if(block_count < 2) block_count = 2;

		BigInteger* out = new BigInteger;
		out->sign = positive;
		if(value == 0) 
			out->sign = zero;
		out->blocks = new unsigned int[block_count];
		memset(out->blocks, 0, block_count*sizeof(unsigned int));

		out->totalBlockCount = block_count;
		out->blocks[0] = value & 0xFFFFFFFF;
		out->blocks[1] = (value >> 32) & 0xFFFFFFFF;
		return out;
	}

	__device__ __host__ static BigInteger* toValue(unsigned __int64 value) {
		return toValue(value, 2);
	}

	__device__ __host__ static BigInteger* toValue(int value, unsigned int block_count) {
		if(block_count < 1) block_count = 1;
		
		BigInteger *out = (BigInteger*)malloc(sizeof(BigInteger));
		out->blocks = (unsigned int*)malloc(block_count*sizeof(unsigned int));
		memset(out->blocks, 0, block_count*sizeof(unsigned int));
		out->sign = positive;
		if(value == 0) 
			out->sign = zero;
		else if(value < 0) {
			out->sign = negative;
			value = -value;
		}
		
		out->totalBlockCount = block_count;
		out->blocks[0] = value;
		return out;
	}

	__device__ __host__ static BigInteger* toValue(int value) {
		return toValue(value, 1);
	}

	__device__ __host__ static BigInteger* toValue(short value, unsigned int block_count) {
		if(block_count < 1) block_count = 1;
		
		BigInteger* out = (BigInteger*)malloc(sizeof(BigInteger));
		out->blocks = (unsigned int*)malloc(block_count*sizeof(unsigned int));
		memset(out->blocks, 0, block_count*sizeof(unsigned int));
		out->sign = positive;
		if(value == 0) 
			out->sign = zero;
		else if(value < 0) {
			out->sign = negative;
			value = -value;
		}
		
		out->totalBlockCount = block_count;
		out->blocks[0] = value;
		return out;
	}

	__device__ __host__ static BigInteger* toValue(short value) {
		return toValue(value, 1);
	}

	__device__ __host__ char toByte() {
		return (char)blocks[0];
	}

	__device__ __host__ unsigned short toUShort() {
		return (unsigned short)blocks[0];
	}

	__device__ __host__ unsigned int toUInt() {
		return blocks[0];
	}
#pragma endregion

#pragma region BigInteger input operations
	__device__ __host__ BigInteger& operator /=(BigInteger &x) {
		Sign thisSign = sign, xSign = x.sign;

		if(thisSign == zero) { } //0 / x = 0 with r of x
		else if(xSign == zero) { //x/0 = undefined. Define it, = 0 with r of this
			x = *this;
			*this = 0;
		}
		else {
			if(thisSign == negative) -(*this);
			if(xSign	== negative) -x;
		
			if(*this < x) {
				x = *this;
				*this = 0;
			}
			else {
				unsigned int shiftCount = 0;
				BigInteger * returnVal = BigInteger::toValue((unsigned __int64)0);
				bool reAdjust = false;
				while(*this > x) {
					reAdjust = true;
					x <<= 1;
					shiftCount++;
				}
			
				if(reAdjust) {
					x >>= 1;
					shiftCount--;
				}

				for(int i = shiftCount; i >= 0; i--) {
					if(*this >= x) {
						*this -= x;
						returnVal->SetBitAtOffset(i);
					}

					x >>= 1;
				}
				BigInteger * returnRemainder = MakeACopy();
				*this = *returnVal;
				free(returnVal->blocks);
				free(returnVal);
				x = *returnRemainder;
				free(returnRemainder->blocks);
				free(returnRemainder);
			}
		}

		return *this;
	}
	
	__device__ __host__ BigInteger& operator *=(BigInteger &x) {
		if(sign == zero) { } //0*x=0
		else if(x.sign == zero) {
			for(int i = 0; i < totalBlockCount; i++) {
				blocks[i] = 0;
			}
		}
		else {
			Sign final_sign = (sign != x.sign) ? negative : positive;

			BigInteger * running_total = BigInteger::toValue((unsigned __int64)0, totalBlockCount);
			for(int i = 0; i < totalBlockCount; i++)
				for(int j = 0; j < x.totalBlockCount; j++) {
					BigInteger * amountToAdd = BigInteger::toValue(((unsigned __int64)blocks[i] * (unsigned __int64)x.blocks[j]));
					*amountToAdd <<= ((i + j) * 32);
					*running_total += *amountToAdd;
					free(amountToAdd->blocks);
					free(amountToAdd);
				}

			running_total->sign = final_sign;

			*this = *running_total;
			free(running_total->blocks);
			free(running_total);
		}

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator |=(BigInteger &x) {
		if(totalBlockCount < x.totalBlockCount) ResizeInteger(x.totalBlockCount);
			
		for(int i = 0; i < x.totalBlockCount; i++)
			blocks[i] |= x.blocks[i];

		//sign bit is ored too
		sign = (x.sign == negative || sign == negative) ? negative : positive;

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator &=(BigInteger &x) {
		 int i = 0;
		for(; i < x.totalBlockCount && i < totalBlockCount; i++)
			blocks[i] &= x.blocks[i];
		for(; i < totalBlockCount; i++)
			blocks[i] = 0;

		//sign bit is ored too
		sign = (x.sign == negative && sign == negative) ? negative : positive;

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator +=(BigInteger &x) {
		if(x.sign == zero) { } //+= 0;, do nothing
		else if(sign == zero) //0 += x;, x
			*this = x;
		else if(sign == x.sign) {
			int k = x.totalBlockCount;
			if(totalBlockCount < x.totalBlockCount) {
				//Get most significant digit
				for(; x.blocks[k - 1] == 0; k--);
				if(k > totalBlockCount) ResizeInteger(k);
			}
			
			unsigned int carryBlock = 0;
			int i = 0;
			for(; i < k; i++) {
				unsigned __int64 thisBlock = (unsigned __int64)carryBlock + (unsigned __int64)blocks[i] + (unsigned __int64)x.blocks[i];
				blocks[i] = (unsigned int)thisBlock;
				carryBlock = (unsigned int)(thisBlock >> 32);				
			}

			if(carryBlock) {
				if(i >= totalBlockCount) { IncreaseIntegerSize(1); }
				blocks[i] += carryBlock;
			}
		}
		else if(sign == negative) {
			BigInteger * buffer = MakeACopy();
			*this = x;
			-(*buffer);
			*this -= *buffer;
			free(buffer->blocks);
			free(buffer);
		}
		else if(x.sign == negative) { //arbatrairy if, this is the only other option
			-x;
			*this -= x;
			-x;
		}

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator -=(BigInteger &x) {
		if(x.sign == zero) { } // -= 0, do nothing
		else if(sign == zero) { //0 -= x, -x
			*this = x;
			-(*this);
		}
		else if(sign == x.sign) {
			if((sign == negative && x < *this) || (sign == positive && *this < x)) {
				BigInteger * buffer = MakeACopy();
				*this = x;
				*this -= *buffer;
				-(*this);
				free(buffer->blocks);
				free(buffer);
			}
			else {
				bool hadToCarry = false;
				for(int i = 0; i < x.totalBlockCount; i++) {
					unsigned __int64 thisBlock = blocks[i];
					if(hadToCarry) thisBlock--;
					hadToCarry = (thisBlock < x.blocks[i]);
					if(hadToCarry) thisBlock += 0x100000000;
					blocks[i] = thisBlock - x.blocks[i];
				}
			}
		}
		else if(sign == negative) {
			-(*this);
			*this += x;
			-(*this);
		}
		else if(x.sign == negative) {
			-x;
			*this += x;
			-x;
		}

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ bool operator <(BigInteger &x) {
		bool returnVal = false;
		if(sign == zero) {
			if(x.sign == positive) returnVal = true;
		}
		else if(x.sign == zero) {
			if(sign == negative) returnVal = true;
		}
		else if(x.sign == sign) {
			int i = (totalBlockCount < x.totalBlockCount) ? x.totalBlockCount-1 : totalBlockCount-1;
			for(; i >= totalBlockCount; i--)
				if(x.blocks[i] != 0) {
					returnVal = (sign == negative) ? false : true;
					goto endThis;
				}
			for(; i >= x.totalBlockCount; i--)
				if(blocks[i] != 0) {
					returnVal = (sign == negative) ? true : false;
					goto endThis;
				}
			for(; i >= 0; i--) {
				if(blocks[i] == x.blocks[i]) continue;
				else if(blocks[i] < x.blocks[i]) { returnVal = (sign == negative) ? false : true; break; }
				else { returnVal = (sign == negative) ? true : false; break; }
			}
		}
		else if(x.sign == positive)
			returnVal = true;
		
		endThis:
		return returnVal;
	}

	__device__ __host__ bool operator >(BigInteger &x) {
		bool returnVal = false;
		if(sign == zero) {
			if(x.sign == positive) returnVal = true;
		}
		else if(x.sign == zero) {
			if(sign == negative) returnVal = false;
		}
		else if(x.sign == sign) {
			int i = (totalBlockCount < x.totalBlockCount) ? x.totalBlockCount-1 : totalBlockCount-1;
			for(; i >= totalBlockCount; i--)
				if(x.blocks[i] != 0) {
					returnVal = (sign == negative) ? true : false;
					goto endThis;
				}
			for(; i >= x.totalBlockCount; i--)
				if(blocks[i] != 0) {
					returnVal = (sign == negative) ? false : true;
					goto endThis;
				}
			for(; i >= 0; i--) {
				if(blocks[i] == x.blocks[i]) continue;
				else if(blocks[i] > x.blocks[i]) { returnVal = (sign == negative) ? false : true; break; }
				else { returnVal = (sign == negative) ? true : false; break; }
			}
		}
		else if(x.sign == positive)
			returnVal = true;
		
		endThis:
		return returnVal;
	}

	__device__ __host__ bool operator >=(BigInteger &x) {
		return !(*this < x);
	}

	__device__ __host__ bool operator <=(BigInteger &x) {
		return !(*this > x);
	}

	__device__ __host__ bool operator ==(BigInteger &x) {
		bool returnVal = true;

		if(x.sign != sign) { returnVal = false; goto endThis; }

		int i = 0;
		for(; i < totalBlockCount && i < x.totalBlockCount; i++)
			if(blocks[i] != x.blocks[i]) { returnVal = false; goto endThis; }

		for(; i < x.totalBlockCount; i++)
			if(0 != x.blocks[i]) { returnVal = false; goto endThis; }

		for(; i < totalBlockCount; i++)
			if(0 != blocks[i]) { returnVal = false; goto endThis; }

		endThis:
		return returnVal;
	}
	
	__device__ __host__ BigInteger& operator =(BigInteger &x) {
		if(totalBlockCount < x.totalBlockCount) {
			free(blocks);
			blocks = (unsigned int*)malloc(x.totalBlockCount * sizeof(unsigned int));
			totalBlockCount = x.totalBlockCount;
		}
		sign = x.sign;
		memset(blocks, 0, totalBlockCount* sizeof(unsigned int));
		memcpy(blocks, x.blocks, x.totalBlockCount * sizeof(unsigned int));
		return *this;
	}
#pragma endregion

#pragma region unsigned __int64 operations
	__device__ __host__ BigInteger& operator %=(unsigned __int64 x) {
		BigInteger * deleteThis = toValue(x);
		*this /= *deleteThis;
		*this = *deleteThis;
		free(deleteThis->blocks);
		free(deleteThis);
		return *this;
	}

	__device__ __host__ BigInteger& operator /=(unsigned __int64 x) {
		BigInteger * deleteThis = toValue(x);
		*this /= *deleteThis;
		free(deleteThis->blocks);
		free(deleteThis);
		return *this;
	}

	__device__ __host__ BigInteger& operator *=(unsigned __int64 x) {
		if(sign == zero) { } //0*x=0
		else if(x == 0) {
			for(int i = 0; i < totalBlockCount; i++) {
				blocks[i] = 0;
			}
		}
		else {
			Sign final_sign = (sign == negative) ? negative : positive;

			BigInteger * running_total = BigInteger::toValue(0, totalBlockCount);
			for(int i = 0; i < totalBlockCount; i++)
				for(int j = 0; j < 2; j++) {
					BigInteger * amountToAdd = BigInteger::toValue((unsigned __int64)blocks[i] * (unsigned __int64)((unsigned int)(x >> (j*32))));
					*amountToAdd <<= ((i + j) * 32);
					*running_total += *amountToAdd;
					free(amountToAdd->blocks);
					free(amountToAdd);
				}

			running_total->sign = final_sign;

			*this = *running_total;
			free(running_total->blocks);
			free(running_total);
		}

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator |=(unsigned __int64 x) {
		if(totalBlockCount < 2) ResizeInteger(2);
			
		for(int i = 0; i < 2; i++)
			blocks[i] |= (x >> (i*32));

		//sign bit is ored too
		sign = (sign == negative) ? negative : positive;

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator &=(unsigned __int64 x) {
		if(totalBlockCount < 2) ResizeInteger(2);
			
		int i = 0;
		for(; i < 2; i++)
			blocks[i] &= (x >> (i*32));
		for(; i < totalBlockCount; i++)
			blocks[i] = 0;

		//sign bit is anded too
		sign = positive;

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator +=(unsigned __int64 x) {
		if(sign == zero) { //0 += x;, x
			if(totalBlockCount < 2) ResizeInteger(2);
			blocks[0] = (unsigned int)x;
			blocks[1] = (unsigned int)(x >> 32);
			sign = positive;
		}
		else if(sign == positive) {
			if(totalBlockCount < 2) ResizeInteger(2);
			
			unsigned int carryBlock = 0;
			for(int i = 0; i < 2; i++) {
				unsigned __int64 thisBlock = (unsigned __int64)carryBlock + (unsigned __int64)blocks[i] + (unsigned __int64)(x >> (i*32));
				blocks[i] = (unsigned int)thisBlock;
				carryBlock = (unsigned int)(thisBlock >> 32);				
			}

			if(carryBlock) {
				IncreaseIntegerSize(1);
				blocks[totalBlockCount-1] = carryBlock;
			}
		}
		else if(sign == negative) {
			BigInteger * buffer = MakeACopy();
			if(totalBlockCount < 2) ResizeInteger(2);
			blocks[0] = (unsigned int)x;
			blocks[1] = (unsigned int)(x >> 32);
			-(*buffer);
			*this -= *buffer;
			free(buffer->blocks);
			free(buffer);
		}

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ BigInteger& operator -=(unsigned __int64 x) {
		if(sign == zero) { //0 -= x, -x
			if(totalBlockCount < 2) ResizeInteger(2);
			blocks[0] = (unsigned int)x;
			blocks[1] = (unsigned int)(x >> 32);
			-(*this);
		}
		else if(sign == positive) {
			if(*this < x) {
				BigInteger * buffer = MakeACopy();
				if(totalBlockCount < 2) ResizeInteger(2);
				blocks[0] = (unsigned int)x;
				blocks[1] = (unsigned int)(x >> 32);
				*this -= *buffer;
				-(*this);
				free(buffer->blocks);
				free(buffer);
			}
			else {
				bool hadToCarry = false;
				for(int i = 1; i >= 0; i--) {
					unsigned __int64 thisBlock = blocks[i];
					if(hadToCarry) thisBlock--;
					hadToCarry = (thisBlock < (x >> (i*32)));
					if(hadToCarry) thisBlock += 0x100000000;
					blocks[i] = thisBlock - (x >> (i*32));
				}
			}
		}
		else if(sign == negative) {
			-(*this);
			*this += x;
			-(*this);
		}

		IsZeroUpdate();
		return *this;
	}

	__device__ __host__ bool operator <(unsigned __int64 x) {
		bool returnVal = false;
		if((sign == zero && x != 0) || sign == negative)
			returnVal = true;
		else if(positive == sign) {
			int i = (totalBlockCount < 2) ? 1 : totalBlockCount-1;
			for(; i >= 2; i--)
				if(blocks[i] != 0) {
					returnVal = false;
					goto endThis;
				}
			for(; i >= totalBlockCount; i--)
				if((unsigned int)(x >> (32*i)) != 0) {
					returnVal = true;
					goto endThis;
				}
			for(; i >= 0; i--) {
				if(blocks[i] == (x >> (32*i))) continue;
				else if(blocks[i] < (x >> (32*i))) { returnVal = true; break; }
				else { returnVal = false; break; }
			}
		}
		
		endThis:
		return returnVal;
	}

	__device__ __host__ bool operator >(unsigned __int64 x) {
		bool returnVal = false;
		if((sign == zero && x != 0) || sign == negative) {
			returnVal = false;
		}
		else if(positive == sign) {
			int i = (totalBlockCount < 2) ? 1 : totalBlockCount-1;
			for(; i >= totalBlockCount; i--)
				if((unsigned int)(x >> (32*i)) != 0) {
					returnVal = false;
					goto endThis;
				}
			for(; i >= 2; i--)
				if(blocks[i] != 0) {
					returnVal = true;
					goto endThis;
				}
			for(; i >= 0; i--) {
				if(blocks[i] == (x >> (32*i))) continue;
				else if(blocks[i] > (x >> (32*i))) { returnVal = true; break; }
				else { returnVal = false; break; }
			}
		}
		
		endThis:
		return returnVal;
	}

	__device__ __host__ bool operator >=(unsigned __int64 x) {
		return !(*this < x);
	}

	__device__ __host__ bool operator <=(unsigned __int64 x) {
		return !(*this > x);
	}

	__device__ __host__ bool operator ==(unsigned __int64 x) {
		bool returnVal = true;

		int i = 0;
		for(; i < totalBlockCount && i < 2; i++)
			if(blocks[i] != (x >> (i*32))) { returnVal = false; goto endThis; }

		for(; i < 2; i++)
			if(0 != (x >> (i*32))) { returnVal = false; goto endThis; }

		for(; i < totalBlockCount; i++)
			if(0 != blocks[i]) { returnVal = false; goto endThis; }

		endThis:
		return returnVal;
	}
	
	__device__ __host__ BigInteger& operator =(unsigned __int64 x) {
		if(totalBlockCount < 2) ResizeInteger(2);
		int i = 0;
		blocks[i++] = (unsigned int)x;
		blocks[i++] = (unsigned int)(x >> 32);

		for(; i < totalBlockCount; i++)
			blocks[i] = 0;

		sign = (x == 0) ? zero : positive;
		return *this;
	}
#pragma endregion
};